#include <stdio.h>
#include <string.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <sstream>
#include <assert.h>
#include <random>
#include <float.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#define uint unsigned int
#define ull unsigned long long

#define DEBUG

__host__ __device__ void Enumerate(
	int root,
	int level,
	int remaining,
	int subgraphSize,
	int** searchTree,
	bool* chosenInTree,
	bool* visitedInCurrentSearch,
	bool** graph,
	int graphSize);

__host__ __device__ void RevolveR(
	int n,
	int left,
	int right,

	int root,
	int level,
	int remaining,
	int subgraphSize,
	int** searchTree,
	bool* chosenInTree,
	bool* visitedInCurrentSearch,
	bool** graph,
	int graphSize);

__host__ __device__ void Revolve(
	int n,
	int left,
	int right,

	int root,
	int level,
	int remaining,
	int subgraphSize,
	int** searchTree,
	bool* chosenInTree,
	bool* visitedInCurrentSearch,
	bool** graph,
	int graphSize)
{
	int* tab = searchTree[level];

	if (n == 0)
	{
		for (int i = left; i < right; ++i)
		{
			chosenInTree[tab[i]] = false;
		}

		Enumerate(
			root,
			level + 1,
			remaining,
			subgraphSize,
			searchTree,
			chosenInTree,
			visitedInCurrentSearch,
			graph,
			graphSize);

	}
	else if (n == right - left)
	{
		for (int i = left; i < right; ++i)
		{
			chosenInTree[tab[i]] = true;
		}

		Enumerate(
			root,
			level + 1,
			remaining,
			subgraphSize,
			searchTree,
			chosenInTree,
			visitedInCurrentSearch,
			graph,
			graphSize);

		for (int i = left; i < right; ++i)
		{
			chosenInTree[tab[i]] = false;
		}
	}
	else
	{
		chosenInTree[tab[left]] = false;
		Revolve(n, left + 1, right,
			root,
			level,
			remaining,
			subgraphSize,
			searchTree,
			chosenInTree,
			visitedInCurrentSearch,
			graph,
			graphSize);

		chosenInTree[tab[left]] = true;
		RevolveR(n - 1, left + 1, right,
			root,
			level,
			remaining,
			subgraphSize,
			searchTree,
			chosenInTree,
			visitedInCurrentSearch,
			graph,
			graphSize);


		chosenInTree[tab[left]] = false;
	}
}

__host__ __device__ void RevolveR(
	int n,
	int left,
	int right,

	int root,
	int level,
	int remaining,
	int subgraphSize,
	int** searchTree,
	bool* chosenInTree,
	bool* visitedInCurrentSearch,
	bool** graph,
	int graphSize)
{
	int* tab = searchTree[level];

	if (n == 0)
	{
		for (int i = left; i < right; ++i)
		{
			chosenInTree[tab[i]] = false;
		}

		Enumerate(
			root,
			level + 1,
			remaining,
			subgraphSize,
			searchTree,
			chosenInTree,
			visitedInCurrentSearch,
			graph,
			graphSize);
	}
	else if (n == right - left)
	{
		for (int i = left; i < right; ++i)
		{
			chosenInTree[tab[i]] = true;
		}

		Enumerate(
			root,
			level + 1,
			remaining,
			subgraphSize,
			searchTree,
			chosenInTree,
			visitedInCurrentSearch,
			graph,
			graphSize);

		for (int i = left; i < right; ++i)
		{
			chosenInTree[tab[i]] = false;
		}
	}
	else
	{
		chosenInTree[tab[right - 1]] = false;
		RevolveR(n, left, right - 1,
			root,
			level,
			remaining,
			subgraphSize,
			searchTree,
			chosenInTree,
			visitedInCurrentSearch,
			graph,
			graphSize);

		chosenInTree[tab[right - 1]] = true;
		Revolve(n - 1, left, right - 1,
			root,
			level,
			remaining,
			subgraphSize,
			searchTree,
			chosenInTree,
			visitedInCurrentSearch,
			graph,
			graphSize);


		chosenInTree[tab[right - 1]] = false;
	}
}

__host__ __device__ void InitChildSet(
	int root,
	int level,
	int** searchTree,
	bool* chosenInTree,
	bool* visitedInCurrentSearch,
	bool** graph,
	int graphSize)
{
	searchTree[level][0] = 0;
	for (int i = 1; i <= searchTree[level - 1][0]; ++i)
	{
		if (chosenInTree[searchTree[level - 1][i]])
		{
			int parent = searchTree[level - 1][i];

			for (int a = root + 1; a < graphSize; ++a)
			{
				if (!visitedInCurrentSearch[a] && a != parent)
				{
					if (graph[parent][a] || graph[a][parent])
					{
						//printf("ADDING %d child of %d \n", a, parent);
						searchTree[level][++searchTree[level][0]] = a;
						visitedInCurrentSearch[a] = true;
					}
				}
			}
		}
	}
}

__host__ __device__ void Enumerate(
	int root,
	int level,
	int remaining,
	int subgraphSize,
	int** searchTree,
	bool* chosenInTree,
	bool* visitedInCurrentSearch,
	bool** graph,
	int graphSize)
{
	if (remaining == 0)
	{
#ifdef DEBUG

#ifdef LEVELS
		for (int lvl = 0; lvl < level; ++lvl)
		{
			printf("%d LEVEL %d:\t", searchTree[lvl][0], lvl);
			for (int i = 1; i <= searchTree[lvl][0]; ++i)
			{
				printf("%d", searchTree[lvl][i] + 1);
			}
			printf("\n");
		}
#endif
		printf("SUBGRAP:\t");
		for (int i = 0; i < graphSize; ++i)
		{
			if (chosenInTree[i]) printf("%d", i + 1);
		}
		printf("\n");
#endif
		return;
	}

	InitChildSet(
		root,
		level,
		searchTree,
		chosenInTree,
		visitedInCurrentSearch,
		graph,
		graphSize);


	for (int k = 1; k <= remaining; ++k)
	{
		if (searchTree[level][0] < k)
		{
			return;
		}

		Revolve(
			k,
			1,
			searchTree[level][0] + 1,
			root,
			level,
			remaining - k,
			subgraphSize,
			searchTree,
			chosenInTree,
			visitedInCurrentSearch,
			graph,
			graphSize);
	}

	for (int i = 1; i <= searchTree[level][0]; ++i)
	{
		visitedInCurrentSearch[searchTree[level][i]] = false;
	}


}

////////////////////////////////////////////////////////////////////////////////
/*
 * 		int root,
		int level,
		int remaining,
		int subgraphSize,
		int** searchTree,
		bool* chosenInTree,
		bool* visitedInCurrentSearch,
		bool** graph,
		int graphSize)
 */
int main(int argc, char** argv)
{
	int root = 0;
	int level = 1;
	int remaring = 3;
	int subgraphSize = 4;
	int** searchTree = new int* [5];

	for (int i = 0; i < 5; ++i)
	{
		searchTree[i] = new int[2000];
	}

	searchTree[0][0] = 1;
	searchTree[0][1] = root;

	bool* chosenInTree = new bool[2000];
	chosenInTree[root] = true;
	bool* visitedInCurrentSearch = new bool[2000];
	bool** graph = new bool* [7];

	for (int i = 0; i < 7; ++i)
	{
		graph[i] = new bool[7];
	}

	graph[0][1] = true;
	graph[0][2] = true;
	graph[1][5] = true;
	graph[2][6] = true;
	graph[2][4] = true;
	graph[2][5] = true;
	graph[3][2] = true;
	graph[4][3] = true;
	graph[4][5] = true;
	graph[4][2] = true;
	graph[4][0] = true;
	graph[5][6] = true;
	graph[5][3] = true;
	graph[6][1] = true;

	int graphSize = 7;

	Enumerate(
		root,
		level,
		remaring,
		subgraphSize,
		searchTree,
		chosenInTree,
		visitedInCurrentSearch,
		graph,
		graphSize);

	printf("HELLO");
}

